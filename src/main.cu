/**
 * Parallel & Distributed Systems
 * Project 4
 * 
 * Ⓒ 2021 K. Chatzis
 * kachatzis <at> ece.auth.gr
 **/

#include <stdio.h>
#include <stdlib.h>
#include <cstdio>
#include <mpi.h>
#include <hip/hip_runtime.h>
#include <omp.h>
#include "utils.hpp"
#include "utils.cuh"
#include "auxlib.hpp"
#include "mmarket.hpp"
#include "mmio.h"
#include "v1.hpp"


using namespace std;


int main(int argc, char** argv)
{

    Runtime rt = startup(argc, argv);

    if(rt.v1){
        v1_execute(rt);
    }

    return 0;
}